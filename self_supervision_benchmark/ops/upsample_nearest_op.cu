#include "hip/hip_runtime.h"
#include "caffe2/core/context_gpu.h"
#include "upsample_nearest_op.h"

// Adapted from https://github.com/torch/cunn/blob/master/lib/THCUNN/SpatialUpSamplingNearest.cu

namespace caffe2 {

namespace {
__device__ int translate_idx(int ii, int d1, int d2, int d3, int scale_factor) {
  int x, y, z, w;
  w = ii % d3;
  ii = ii/d3;
  z = ii % d2;
  ii = ii/d2;
  y = ii % d1;
  ii = ii/d1;
  x = ii;
  w = w/scale_factor;
  z = z/scale_factor;
  d2 /= scale_factor;
  d3 /= scale_factor;
  return (((x*d1+y)*d2)+z)*d3+w;
}

__device__ int translate_idx_inv(
    int ii, int d1, int d2, int d3, int scale_factor, int off_x, int off_y) {
  int x, y, z, w;
  w = ii % d3;
  ii = ii/d3;
  z = ii % d2;
  ii = ii/d2;
  y = ii % d1;
  ii = ii/d1;
  x = ii;
  w = w*scale_factor+off_x;
  z = z*scale_factor+off_y;
  d2 *= scale_factor;
  d3 *= scale_factor;
  return (((x*d1+y)*d2)+z)*d3+w;
}

__global__ void upscale(const float *input, float *output, long no_elements,
                        int scale_factor, int d1, int d2, int d3) {
  long ii = threadIdx.x + blockDim.x * blockIdx.x;
  ii += threadIdx.y + blockDim.y * (blockDim.x * gridDim.x) * blockIdx.y;
  if (ii >= no_elements) return;
  int ipidx = translate_idx(ii, d1, d2, d3, scale_factor);
  output[ii]=input[ipidx];
}

__global__ void downscale(float *gradInput_data, const float *gradOutput_data,
                          long no_elements, int scale_factor, int d1, int d2,
                          int d3) {
  long ii = threadIdx.x + blockDim.x * blockIdx.x;
  ii += threadIdx.y + blockDim.y * (blockDim.x * gridDim.x) * blockIdx.y;
  if (ii >= no_elements) return;
  for (int i=0; i < scale_factor; i++){
    for(int j=0; j < scale_factor; j++){
      int ipidx = translate_idx_inv(ii, d1, d2, d3, scale_factor, i, j);
      gradInput_data[ii] += gradOutput_data[ipidx];
    }
  }
}
} // namespace

template<>
bool UpsampleNearestOp<float, HIPContext>::RunOnDevice() {
  auto& X = Input(0);
  auto* Y = Output(0);

  vector<int64_t> out_shape;
  for (int i = 0; i < X.dim(); ++i) {
    out_shape.push_back(X.dim32(i));
  }
  out_shape[X.dim() - 1] *= scale_;
  out_shape[X.dim() - 2] *= scale_;
  Y->Resize(out_shape);

  int d1;
  int d2;
  int d3;
  if (X.dim() == 3) {
    d1 = Y->dim32(0);
    d2 = Y->dim32(1);
    d3 = Y->dim32(2);
  } else {
    d1 = Y->dim32(1);
    d2 = Y->dim32(2);
    d3 = Y->dim32(3);
  }
  long no_elements = Y->size();

  const float *input_data = X.data<float>();
  float *output_data = Y->mutable_data<float>();

  // cuda blocks & threads:
  long nthreads = 256;
  // Max number of blocks: http://en.wikipedia.org/wiki/CUDA
  // 65535 for SM 2.x, 2^32 -1 for >= 3.0
  // TODO: When we move to SM 3.5 we should update this
  long n_xblocks = min(max((int)ceil((float)no_elements / nthreads), 1), 65535);
  long n_yblocks = (long)ceil(
      (float)no_elements / (float)(n_xblocks * nthreads));
  CAFFE_ENFORCE(n_yblocks <= 65535);
  dim3 blocks(n_xblocks, n_yblocks);
  dim3 threads(nthreads);

  upscale<<<blocks, threads, 0, context_.hip_stream()>>>(
      input_data, output_data, no_elements, scale_, d1, d2, d3);
  return true;
}


template<>
bool UpsampleNearestGradientOp<float, HIPContext>::RunOnDevice() {
  auto& X  = Input(0);   // Original input to "forward" op
  auto& dY = Input(1);   // Gradient of net w.r.t. output of "forward" op
                         // (aka "gradOutput")
  // auto* dX = Output(0);  // Gradient of net w.r.t. input to "forward" op
                         // (aka "gradInput")

  // dX->ResizeLike(X);
  auto* dX = Output(0, X.sizes(), at::dtype<float>());
  float *gradInput_data = dX->mutable_data<float>();
  const float *gradOutput_data = dY.data<float>();

  int d1;
  int d2;
  int d3;
  if (dX->ndim() == 3) {
    d1 = dX->dim32(0);
    d2 = dX->dim32(1);
    d3 = dX->dim32(2);
  } else {
    d1 = dX->dim32(1);
    d2 = dX->dim32(2);
    d3 = dX->dim32(3);
  }
  long no_elements = dX->size();

  // cuda blocks & threads:
  long nthreads = 256;
  // Max number of blocks: http://en.wikipedia.org/wiki/CUDA
  // 65535 for SM 2.x, 2^32 -1 for >= 3.0
  // TODO: When we move to SM 3.5 we should update this
  long n_xblocks = min(max((int)ceil((float)no_elements / nthreads), 1), 65535);
  long n_yblocks = (long)ceil(
      (float)no_elements / (float)(n_xblocks * nthreads));
  CAFFE_ENFORCE(n_yblocks <= 65535);
  dim3 blocks(n_xblocks, n_yblocks);
  dim3 threads(nthreads);

  math::Set<float, HIPContext>(no_elements, 0.f, gradInput_data, &context_);
  downscale<<<blocks, threads, 0, context_.hip_stream()>>>(
      gradInput_data, gradOutput_data, no_elements, scale_, d1, d2, d3);

  return true;
}

REGISTER_HIP_OPERATOR(UpsampleNearest,
                       UpsampleNearestOp<float, HIPContext>);
REGISTER_HIP_OPERATOR(UpsampleNearestGradient,
                       UpsampleNearestGradientOp<float, HIPContext>);
} // namespace caffe2
